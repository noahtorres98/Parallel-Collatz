/*
Collatz code for CS 4380 / CS 5351

Copyright (c) 2020 Texas State University. All rights reserved.

Redistribution in source or binary form, with or without modification,
is *not* permitted. Use in source or binary form, with or without
modification, is only permitted for academic use in CS 4380 or CS 5351
at Texas State University.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

Author: Martin Burtscher
*/

#include <cstdio>
#include <algorithm>
#include <sys/time.h>
#include <hip/hip_runtime.h>

static const int ThreadsPerBlock = 1024;

static __global__ void collatz(int *maxlen)
{
	// compute sequence lengths
	const long i = 1 + 2 *(threadIdx.x + blockIdx.x * (long)blockDim.x);
	long val = i;
	int len = 1;
	while (val != 1) {
		len++;
		if ((val % 2) == 0) {
			val /= 2;  // even
		} else {
			val = 3 * val + 1;  // odd
		}
	}

	if (len > *maxlen){
		atomicMax(maxlen, len);
	}
}

static void CheckCuda()
{
  hipError_t e;
  hipDeviceSynchronize();
  if (hipSuccess != (e = hipGetLastError())) {
    fprintf(stderr, "CUDA error %d: %s\n", e, hipGetErrorString(e));
    exit(-1);
  }
}

int main(int argc, char *argv[])
{
  printf("Collatz v1.4\n");

  // check command line
  if (argc != 2) {fprintf(stderr, "USAGE: %s upper_bound\n", argv[0]); exit(-1);}
  const long bound = atol(argv[1]);
  if (bound < 1) {fprintf(stderr, "ERROR: upper_bound must be at least 1\n"); exit(-1);}
  printf("upper bound: %ld\n", bound);

  // start time
  timeval start, end;
  gettimeofday(&start, NULL);

  // execute timed code
  int *device_maxlen;
  int host_maxlen;
  int size = sizeof(int);

  if (hipSuccess != hipMalloc((void **) &device_maxlen, size)) {fprintf(stderr, "ERROR could not allocate mem 1st\n"); exit(-1);}

  host_maxlen = 0;

  if (hipSuccess != hipMemcpy(device_maxlen, &host_maxlen, size, hipMemcpyHostToDevice)) {fprintf(stderr, "ERROR could not allocate mem 2nd\n"); exit(-1);}

  gettimeofday(&start, NULL);

  collatz<<<((bound + 1)/ 2 + ThreadsPerBlock - 1) / ThreadsPerBlock, ThreadsPerBlock>>>(device_maxlen);
  hipDeviceSynchronize();

  // end time
  gettimeofday(&end, NULL);
  const double runtime = end.tv_sec - start.tv_sec + (end.tv_usec - start.tv_usec) / 1000000.0;
  printf("compute time: %.6f s\n", runtime);

  CheckCuda();

  if (hipSuccess != hipMemcpy(&host_maxlen, device_maxlen, size, hipMemcpyDeviceToHost)) {fprintf(stderr, "ERROR could not allocate mem 3rd\n"); exit(-1);}

  // print result
  printf("longest sequence length: %d elements\n", host_maxlen);
  return 0;
}
